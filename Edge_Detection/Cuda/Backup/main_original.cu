//============================================================================
// Name        : main.cu
// Author      : Daniele Gadler
// Version     :
// Description : Sobel operator in CUDA
//============================================================================


#include <stdio.h>


#include "functions.c"


//false --> No vertical gradient and horizontal gradient are output
//true --> Vertical gradient and horizontal gradient are output
#define INTERMEDIATE_OUTPUT false
#define SOBEL_OP_SIZE 9
#define STRING_BUFFER_SIZE 1024

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
#define get_time(time) (gettimeofday(&time, NULL))


#include "string.h"
#include "stdlib.h"
#include "math.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#include "kernels.cu"


static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE);
    }
}



int main ( int argc, char** argv )
{
		//all the time_val declarations are put at the beginning of the file for better code readability
		struct timeval comp_start_load_img, comp_end_load_img;
		struct timeval i_o_start_load_img, i_o_end_load_img;
		struct timeval comp_start_img_conv, comp_end_img_conv;
		struct timeval start_alloc_rgb, end_alloc_rgb;
		struct timeval start_first_cuda_malloc, end_first_cuda_malloc;
		struct timeval comp_start_rgb_to_gray, comp_end_rgb_to_gray;
		struct timeval start_gray_vec_copy, end_gray_vec_copy;
		struct timeval comp_start_str_alloc, comp_end_str_alloc;
		struct timeval start_free_rgb, end_free_rgb;
		struct timeval comp_start_alloc_h_vec, comp_end_alloc_h_vec;
		struct timeval start_h_vec_alloc, end_h_vec_alloc;
		struct timeval comp_start_horiz_grad, comp_end_horiz_grad;
		struct timeval start_h_vec_copy, end_h_vec_copy;
		struct timeval start_h_vec_free, end_h_vec_free;
		struct timeval comp_start_alloc_v_grad, comp_end_alloc_v_grad;
		struct timeval start_v_vec_alloc, end_v_vec_alloc;
		struct timeval comp_start_vert_grad, comp_end_vert_grad;
		struct timeval start_v_vec_copy, end_v_vec_copy;
		struct timeval start_countour_alloc, end_countour_alloc;
		struct timeval start_countour_copy, end_countour_copy;
		struct timeval start_free_countour, end_free_countour;
		struct timeval i_o_start_write_img, i_o_end_write_img;

		//dummy CUDA malloc to "waste" time just at the beginning of the program and not in the middle of the computation
		byte * dummy_array;
		get_time(start_first_cuda_malloc);
		HANDLE_ERROR ( hipMalloc((void **)&dummy_array , 1*sizeof(byte)));
	    get_time(end_first_cuda_malloc);

		//actual computation begins
		get_time(comp_start_load_img);
		if(argc < 2)
		{
			printf("You did not provide any input image name. Please, provide an input image name and retry. \n");
			return -2;
		}

		//###########1. STEP - LOAD THE IMAGE, ITS HEIGHT, WIDTH AND CONVERT IT TO RGB FORMAT#########

		//Specify the input image. Formats supported: png, jpg, GIF.
		const char * file_output_rgb = "imgs_out/image.rgb";
		const char *png_strings[4] = {"convert ", argv[1], " ", file_output_rgb};
		const char * str_PNG_to_RGB = array_strings_to_string(png_strings, 4, STRING_BUFFER_SIZE);

		//printf("Loading input image [%s] \n", fileInputName); //debug

		get_time(comp_end_load_img);

		get_time(i_o_start_load_img);
		//execute the conversion from PNG to RGB, as that format is required by the program
		int status_conversion = system(str_PNG_to_RGB);
		get_time(i_o_end_load_img);

		get_time(comp_start_img_conv);
		if(status_conversion != 0)
		{
			printf("ERROR! Conversion of input PNG image to RGB was not successful. Program aborting.\n");
			return -1;
		}
		//get the height and width of the input image
		int width = 0;
		int height = 0;

		get_image_size(argv[1], &width, &height);

		//Three dimensions because the input image is in RGB format
		int rgb_size = width * height * 3;

		//Used as a buffer for all pixels of the image
		byte * rgb_image;

		//Load up the input image in RGB format into one single flattened array (rgbImage)
		read_file(file_output_rgb, &rgb_image, rgb_size);

		//########2. step - convert RGB image to gray-scale
	    int gray_size = rgb_size / 3;
	    byte * r_vector, * g_vector, * b_vector;

	    //now take the RGB image vector and create three separate arrays for the R,G,B dimensions
	    get_dimension_from_RGB_vec(0, rgb_image,  &r_vector, gray_size);
	    get_dimension_from_RGB_vec(1, rgb_image,  &g_vector, gray_size);
	    get_dimension_from_RGB_vec(2, rgb_image,  &b_vector, gray_size);

	    //allocate memory on the device for the r,g,b vectors
	    byte * dev_r_vec, * dev_g_vec, * dev_b_vec;
	    byte * dev_gray_image;

		get_time(comp_end_img_conv);
		get_time(start_alloc_rgb);

	    HANDLE_ERROR ( hipMalloc((void **)&dev_r_vec, gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_g_vec, gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_b_vec, gray_size*sizeof(byte)));

	    //copy the content of the r,g,b vectors from the host to the device
	    HANDLE_ERROR (hipMemcpy (dev_r_vec , r_vector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_g_vec , g_vector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_b_vec , b_vector, gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    //allocate memory on the device for the output gray image
	    HANDLE_ERROR ( hipMalloc((void **)&dev_gray_image, gray_size*sizeof(byte)));

		get_time(end_alloc_rgb);

		get_time(comp_start_rgb_to_gray);

	    //actually run the kernel to convert input RGB file to gray-scale
	    rgb_img_to_gray <<< width, height>>> (dev_r_vec, dev_g_vec, dev_b_vec, dev_gray_image, gray_size) ;
	    hipDeviceSynchronize();

		byte * gray_image = (byte *) malloc(gray_size * sizeof(byte));

		get_time(comp_end_rgb_to_gray);

		get_time(start_gray_vec_copy);
	    //Now take the device gray vector and bring it back to the host
	    HANDLE_ERROR (hipMemcpy(gray_image , dev_gray_image , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		get_time(end_gray_vec_copy);

		get_time(comp_start_str_alloc);
		char str_width[100];
		sprintf(str_width, "%d", width);

		char str_height[100];
		sprintf(str_height, "%d", height);

		get_time(comp_end_str_alloc);

		//output the gray-scale image to a PNG file if INTERMEDIATE_OUTPUT == true
		output_gray_scale_image(INTERMEDIATE_OUTPUT, gray_image, gray_size, str_width, str_height, STRING_BUFFER_SIZE, "imgs_out/img_gray.png");

		get_time(start_free_rgb);
	    hipFree (dev_r_vec);
	    hipFree (dev_g_vec);
		hipFree (dev_b_vec);
		get_time(end_free_rgb);

		//######################3. Step - Compute vertical and horizontal gradient ##########

		//###Compute the HORIZONTAL GRADIENT#####

		get_time(comp_start_alloc_h_vec);
   	    //host horizontal kernel
		int sobel_h[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
		int * dev_sobel_h;
   	    byte * dev_sobel_h_res;
		get_time(comp_end_alloc_h_vec);

		get_time(start_h_vec_alloc);

		//allocate memory for device horizontal kernel
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h , SOBEL_OP_SIZE*sizeof(int)));

		//copy the content of the host horizontal kernel to the device horizontal kernel
	    HANDLE_ERROR (hipMemcpy (dev_sobel_h , sobel_h , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));

	    //allocate memory for the resulting horizontal gradient on the device
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h_res , gray_size*sizeof(byte)));

		get_time(end_h_vec_alloc);

		get_time(comp_start_horiz_grad);
		//perform horizontal gradient calculation for every pixel
		it_conv <<< width, height>>> (dev_gray_image, gray_size, width, dev_sobel_h, dev_sobel_h_res);
	    hipDeviceSynchronize();

		//fixed segmentation fault when processing large images by using a malloc
		byte* sobel_h_res = (byte*) malloc(gray_size * sizeof(byte));
		get_time(comp_end_horiz_grad);
		//copy the resulting horizontal array from device to host

		get_time(start_h_vec_copy);
	    HANDLE_ERROR (hipMemcpy(sobel_h_res , dev_sobel_h_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
	    get_time(end_h_vec_copy);


		get_time(start_h_vec_free);
	    //free-up the memory for the vectors allocated
	    hipFree(dev_sobel_h);
	    get_time(end_h_vec_free);

	    //output the horizontal gradient to a file if INTERMEDIATE_OUTPUT == true
	    output_gradient(INTERMEDIATE_OUTPUT, sobel_h_res, gray_size, str_width, str_height, STRING_BUFFER_SIZE, "imgs_out/sobel_horiz_grad.png");

		get_time(comp_start_alloc_v_grad);
		//####Compute the VERTICAL GRADIENT#####
	    int sobel_v[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
		int * dev_sobel_v;
		byte * dev_sobel_v_res;
		get_time(comp_end_alloc_v_grad);

		get_time(start_v_vec_alloc);

		//allocate memory for device vertical kernel
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v , SOBEL_OP_SIZE*sizeof(int)));

		//copy the content of the host vertical kernel to the device vertical kernel
		HANDLE_ERROR (hipMemcpy (dev_sobel_v , sobel_v , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));

		//allocate memory for the resulting vertical gradient on the device
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v_res , gray_size*sizeof(byte)));

		get_time(end_v_vec_alloc);

		get_time(comp_start_vert_grad);

		//perform vertical gradient calculation for every pixel
		it_conv <<<width, height>>> (dev_gray_image, gray_size, width, dev_sobel_v, dev_sobel_v_res);
	    hipDeviceSynchronize();

		//copy the resulting vertical array from device back to host
		//fixed segmentation fault issue with big images
		byte* sobel_v_res = (byte*) malloc(gray_size * sizeof(byte));

		get_time(comp_end_vert_grad);


		get_time(start_v_vec_copy);
		HANDLE_ERROR (hipMemcpy(sobel_v_res , dev_sobel_v_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		get_time(end_v_vec_copy);

		//free-up the memory for the vectors allocated
		struct timeval start_v_vec_free, end_v_vec_free;
		get_time(start_v_vec_free);
		hipFree(dev_sobel_v);
		get_time(end_v_vec_free);

		struct timeval comp_start_countour_alloc, comp_end_countour_alloc;
		get_time(comp_start_countour_alloc);

	    output_gradient(INTERMEDIATE_OUTPUT, sobel_v_res, gray_size, str_width, str_height, STRING_BUFFER_SIZE, "imgs_out/sobel_vert_grad.png");

		//#############4. Step - Compute the countour by putting together the vertical and horizontal gradients####
		//allocate device memory for the final vector containing the countour

		byte * dev_countour_img;
		get_time(comp_end_countour_alloc);


		get_time(start_countour_alloc);
		HANDLE_ERROR ( hipMalloc((void **)&dev_countour_img , gray_size*sizeof(byte)));
		get_time(end_countour_alloc);

		struct timeval comp_start_countour_merge, comp_end_countour_merge;

		get_time(comp_start_countour_merge);
		contour <<< width, height>>> (dev_sobel_h_res, dev_sobel_v_res, gray_size, dev_countour_img);
	    hipDeviceSynchronize();

		//copy the resulting countour image from the device back to host
		byte * countour_img = (byte *) malloc(gray_size * sizeof(byte));

		get_time(comp_end_countour_merge);

		get_time(start_countour_copy);
		HANDLE_ERROR (hipMemcpy(countour_img, dev_countour_img, gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		get_time(end_countour_copy);

		get_time(start_free_countour);
		//free-up all the memory from the allocated vectors
	    hipFree(dev_sobel_h_res);
	    hipFree(dev_sobel_v_res);
	    hipFree(dev_countour_img);
	    get_time(end_free_countour);

	    //######Display the resulting countour image

		get_time(i_o_start_write_img);
	    output_gradient(true, countour_img, gray_size, str_width, str_height, STRING_BUFFER_SIZE, "imgs_out/sobel_countour.png");
		get_time(i_o_end_write_img);


		//#############5. Step - Display the elapsed time in the different parts of the code

		//##GPU memory movements (hipMalloc, cudaMemCpy, hipFree) ##
		double time_alloc_rgb = compute_elapsed_time(start_alloc_rgb, end_alloc_rgb);
		double time_free_rgb = compute_elapsed_time(start_free_rgb, end_free_rgb);
		double time_copy_gray = compute_elapsed_time(start_gray_vec_copy, end_gray_vec_copy);
		//horizontal vector operations
		double time_alloc_h_vec = compute_elapsed_time(start_h_vec_alloc, end_h_vec_alloc);
		double time_copy_h_vec = compute_elapsed_time(start_h_vec_copy, end_h_vec_copy);
		double time_free_h_vec = compute_elapsed_time(start_h_vec_free, end_h_vec_free);
		//vertical vector operations
		double time_alloc_v_vec = compute_elapsed_time(start_v_vec_alloc, end_v_vec_alloc);
		double time_copy_v_vec = compute_elapsed_time(start_v_vec_copy, end_v_vec_copy);
		double time_free_v_vec = compute_elapsed_time(start_v_vec_free, end_v_vec_free);
		//countour image operations
		double time_alloc_countour = compute_elapsed_time(start_countour_alloc, end_countour_alloc);
		double time_copy_countour = compute_elapsed_time(start_countour_copy, end_countour_copy);
		double time_free_countour = compute_elapsed_time(start_free_countour, end_free_countour);

		double total_time_gpu_mem = time_alloc_rgb + time_free_rgb + time_copy_gray + time_alloc_h_vec + time_copy_h_vec + time_free_h_vec +
							  time_alloc_v_vec + time_copy_v_vec + time_free_v_vec + time_alloc_countour + time_copy_countour + time_free_countour;

		printf("Time spent on GPU memory operations: [%f] ms\n", total_time_gpu_mem); //debug
		//printf("%f \n", total_time_gpu_mem);


		//##Actual GPU computation##
		double comp_time_load_img = compute_elapsed_time(comp_start_load_img, comp_end_load_img);
		double comp_time_convert_img = compute_elapsed_time(comp_start_img_conv, comp_end_img_conv);
		double comp_time_rgb_to_gray = compute_elapsed_time(comp_start_rgb_to_gray, comp_end_rgb_to_gray);
		double comp_time_str_alloc = compute_elapsed_time(comp_start_str_alloc, comp_end_str_alloc);
		double comp_time_h_alloc = compute_elapsed_time(comp_start_alloc_h_vec, comp_end_alloc_h_vec);
		double comp_time_h_grad = compute_elapsed_time(comp_start_horiz_grad, comp_end_horiz_grad);
		double comp_time_v_alloc = compute_elapsed_time(comp_start_alloc_v_grad, comp_end_alloc_v_grad);
		double comp_time_v_grad = compute_elapsed_time(comp_start_vert_grad, comp_end_vert_grad);
		double comp_time_count_alloc = compute_elapsed_time(comp_start_countour_alloc, comp_end_countour_alloc);
		double comp_time_count_merge = compute_elapsed_time(comp_start_countour_merge, comp_end_countour_merge);

		double total_time_gpu_comp = comp_time_load_img + comp_time_convert_img + comp_time_rgb_to_gray + comp_time_str_alloc + comp_time_h_alloc +
						comp_time_h_grad + comp_time_v_alloc + comp_time_v_grad + comp_time_count_alloc + comp_time_count_merge ;

		printf("Time spent on GPU computation: [%f] ms\n", total_time_gpu_comp); //debug
		//printf("%f \n", total_time_gpu_comp);

		//##Input/Output over the disk (image loading and final image writing)##
		double i_o_time_load_img = compute_elapsed_time(i_o_start_load_img, i_o_end_load_img);
		double i_o_time_write_img = compute_elapsed_time(i_o_start_write_img, i_o_end_write_img);

		double total_time_i_o = i_o_time_load_img + i_o_time_write_img;

		printf("Time spent on I/O operations from/to disk: [%f] ms\n", total_time_i_o); //debug
		//printf("%f \n", total_time_i_o);

		//##Overall time spent in the program
		double overall_total_time = total_time_gpu_comp + total_time_gpu_mem + total_time_i_o;

		printf("Overall time spent in program [%f] ms \n", overall_total_time); //debug
		//printf("%f \n", overall_total_time);


		double time_first_cuda_malloc = compute_elapsed_time(start_first_cuda_malloc, end_first_cuda_malloc);

		//printf("First cuda malloc has taken [%f] ms\n", time_first_cuda_malloc);

		//let's deallocate the heap memory to avoid any memory leaks
		free(gray_image);
		free(sobel_h_res);
		free(sobel_v_res);
		free(countour_img);

	    return 0;

}
